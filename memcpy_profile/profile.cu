#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
#include <string>
#include <vector>
#include <fstream>

long avgTime(std::vector<long> times) {
  long long total = 0;
  for (double t : times) {
    total += t;
  }
  return total / times.size();
}

std::vector<int> readLayerSize(std::string logpath) {
  std::ifstream infile(logpath);
  if(!infile.good()) {
    std::cout << "open file " << logpath << "error\n";
    std::vector<int> empty;
    return empty;
  }
  std::string line;
  std::vector<int> sizes;
  while (std::getline(infile, line)) {
    sizes.push_back(std::stoi(line));
  }
  return sizes;
}

int main(int argc, char *argv[]) {
  if (argc < 3) {
    std::cerr << "Usage" << argv[0] << "<size> <repeat-times>\n";
    return 1;
  }
  std::string arg1 = argv[1];
  std::string arg2 = argv[2];
  const unsigned int N = std::stoi(arg1);
  const unsigned int bytes = N * sizeof(float);
  std::cout << "transfer data size: " << bytes << " bytes" << std::endl;
  int *h_a = (int *)malloc(bytes);
  int *d_a;
  hipMalloc((int **)&d_a, bytes);

  memset(h_a, 0, bytes);
  std::vector<long> h2d_times;
  std::vector<long> d2h_times;
  for (int i = 0; i < stoi(arg2); i++) {
    auto s = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    auto e = std::chrono::high_resolution_clock::now();
    h2d_times.push_back((e - s).count());

    s = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    e = std::chrono::high_resolution_clock::now();
    d2h_times.push_back((e - s).count());
  }

  std::cout << "Host to Device memcopy " << avgTime(h2d_times) << " ns\n";
  std::cout << "Device to Host " << avgTime(d2h_times) << " ns\n";

  return 0;
}